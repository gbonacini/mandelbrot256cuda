#include "hip/hip_runtime.h"
// -------------------------------------------------------------------------
// mandelbrot256cuda - a command line program able to render Mandelbrot set 
//                on terminal supporting ANSI ecape sequences.
// Copyright (C) 2024  Gabriele Bonacini
//
// This program is free software for no profit use, then you can redistribute 
// it and/or modify it under the terms of the GNU General Public License 
// as published by the Free Software Foundation; either version 2 of 
// the License, or (at your option) any later version.
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software Foundation,
// Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301  USA
// A commercial license is also available for a lucrative use.
// -------------------------------------------------------------------------

 #include <iostream>
 #include <algorithm>
 #include <random>
 #include <chrono>
 #include <cstdlib>

 #include <hip/hip_runtime.h>
 #include <math.h>

 #include <mandelbrot256cuda.hpp>

namespace mandelbrot256cuda {

 using std::chrono::system_clock,
       std::default_random_engine,
       std::shuffle,
       std::cout,
       std::cerr,
       std::abort;

  __global__ void cudaRender(int* out, int pixels, int width, int height, int maxiter, Complex span, Complex begin ){
     auto calculate { [](Complex cmp, int maxiter) -> int {
                           int  n { 0 };
                           for(Complex z { cmp }; n < maxiter; n++) {
			       double absz { sqrtf( z.real() * z.real() + z.imag() * z.imag()) };
                               if( absz >= 2.0) break;
                               z = z * z + cmp;
                           }
                           return n == maxiter ? 0 : n;
                      }
                     };

     unsigned int idx { blockIdx.x * blockDim.x + threadIdx.x },
                  blk { blockDim.x * gridDim.x };

     for(unsigned int pix { idx } ; pix < pixels; pix+= blk) {
         const unsigned int x { pix % width }, 
                            y { pix / width };
         
         Complex   c { begin + Complex(x * span.real() / (width + 1.0),
                       y * span.imag() / (height + 1.0)) };
         
	 out[pix] = calculate(c, maxiter);
     }

 }
       
 
 MandelbrotShell::MandelbrotShell(int w, int h, int z, int dx, int dy, int max, int blocks)  noexcept
     : width{w}, height{h}, zoom{z}, deltax{ dx }, deltay{ dy }, pixels{w * h}, maxiter{max}, cudaBlocks{blocks}, 
       center(-0.8 + deltax / 10.0, 0.0 + deltay / 10.0), span(2.7/zoom, -(5/3.0)*2.7*height/width/zoom), begin { center - span/2.0}
 {

    int cudaDetectedDevices  { 0 };
    hipError_t errorId      { hipGetDeviceCount(&cudaDetectedDevices) };
    if(errorId != hipSuccess) {
	cerr << "Error probing Cuda devices: " << errorId << " - " << hipGetErrorString(errorId) << "\n";
	abort();
    }

    if (cudaDetectedDevices == 0) {
	cerr << "Error: No Cuda device found\n";
	abort();
    }

    if(hipMallocManaged(&out, pixels * sizeof(int)) != hipSuccess){
	cerr << "Error: allocating unified memory\n";
	abort();
    }
 }

 MandelbrotShell::~MandelbrotShell(void)  noexcept{
    hipFree(out);
 }

 void MandelbrotShell::setFullColours(void) noexcept {
     long seed { system_clock::now().time_since_epoch().count() };
     default_random_engine e(seed);
     palette.resize(256);
     for(unsigned int i{0x0}; i<= 0xFF; i++)
        palette[i]=static_cast<unsigned char>(i);

     shuffle(palette.begin(), palette.end(), e);     
 }

 void  MandelbrotShell::setWidth(int ww) noexcept{
     width=ww;
 }

 void  MandelbrotShell::setHeight(int hh) noexcept{
     height=hh;
 }

 void  MandelbrotShell::setMaxiter(int max) noexcept{
     maxiter=max;
 }

 void  MandelbrotShell::setZoom(int zz) noexcept{
     zoom=zz;
 }

 void  MandelbrotShell::setDeltaX(int dx) noexcept{
     deltax=dx;
 }

 void  MandelbrotShell::setDeltaY(int dy) noexcept{
     deltay=dy;
 }

 void MandelbrotShell::setCustomPalette(PaletteInit init)  noexcept{
     if(init.size() >= MIN_PALETTE_LEN && init.size() <= 0xFF + 1) palette = init;
 }

 void MandelbrotShell::render(void) const noexcept{
     int blockSize { cudaBlocks },
         numBlocks { (pixels + blockSize - 1) / blockSize };
     cudaRender<<<numBlocks, blockSize>>>(out, pixels, width, height, maxiter, span, begin);
     hipDeviceSynchronize();
 }

 void MandelbrotShell::print(void) const noexcept{
     for(int pix = 0 ; pix < pixels; ++pix) {
           unsigned char colour { out[pix]  > 0 ? palette[out[pix]  % (palette.size() - 1)] : static_cast<unsigned char>(0x00) };
           const    int  x { pix % width }; 

           cout << "\033[48;5;" << static_cast<int>(colour) << "m \033[m";
           if(x + 1 == width) cout << '\n';
     }
 }

 } //End Namespace
